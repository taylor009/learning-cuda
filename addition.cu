#include <stdio.h>

#include <hip/hip_runtime.h>

// CUDA kernel function to add two numbers
__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

// Function to perform addition using CUDA
void performAddition(int a, int b, int *c) {
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1, 1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}