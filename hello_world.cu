
#include <hip/hip_runtime.h>
#include <stdio.h>

// Declare the function defined in addition.cu
void performAddition(int a, int b, int *c);

void vectorAddition();

int main(void) {
    int a = 3;
    int b = 7;
    int c;

    // Perform addition using CUDA
    // performAddition(a, b, &c);
    vectorAddition();

    // Print the result
    printf("%d\n", c);

    return 0;
}