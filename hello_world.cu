
#include <hip/hip_runtime.h>
#include <stdio.h>

// Declare the function defined in addition.cu
void performAddition(int a, int b, int *c);

int main(void) {
    int a = 3;
    int b = 7;
    int c;

    // Perform addition using CUDA
    performAddition(a, b, &c);

    // Print the result
    printf("%d\n", c);

    return 0;
}