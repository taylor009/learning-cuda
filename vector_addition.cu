#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>

// block size
#define BLOCK_SIZE 512
#define VECTOR_SIZE 100 // We can change the value of W to 200, 400, 800, 1600, 3200

// Allocates a vector with random float entries

void randomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
    {
        data[i] = rand() / (float)RAND_MAX;
    }
}

__global__ void VectorAdd(float* Md, float* Nd, float* Pd) {
    // Calculate the index of the Pd element and M and N
    int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // Each thread computes one element of the block sub-vector
    Pd[index] = Md[index] + Nd[index];
}

void vectorAddition() {
   srand(2006);
    double time;
    clock_t stime = clock();
    clock_t etime;

    // Allocate host memory for vectors A and B
    unsigned int size_A = VECTOR_SIZE;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*)malloc(mem_size_A);

    unsigned int size_B = VECTOR_SIZE;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*)malloc(mem_size_B);

    // Initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    hipMalloc((void**)&d_A, mem_size_A);

    float* d_B;
    hipMalloc((void**)&d_B, mem_size_B);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);

    // allocate device memory for result
    unsigned int size_C = VECTOR_SIZE;
    unsigned int mem_size_C = sizeof(float) * size_C;

    float* d_C;
    hipMalloc((void**)&d_C, mem_size_C);

    // setup execution parameters
    dim3 threads(1, BLOCK_SIZE);
    dim3 grid(VECTOR_SIZE / threads.x, threads.y);

    // execute the kernal
    VectorAdd <<< grid, threads >>> (d_A, d_B, d_C);

    hipDeviceSynchronize();

    // allocate host memory for the result
    float* h_C = (float*)malloc(mem_size_C);

    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);

    // cleanup memory
    free(h_A);
    free(h_B);
    free(h_C);
}